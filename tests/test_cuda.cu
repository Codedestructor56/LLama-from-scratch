#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

// Matrix multiplication kernel
__global__ void matmul(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < N && col < N) {
        float sum = 0.0f;
        for (int i = 0; i < N; ++i) {
            sum += A[row * N + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}

// Host function to test the kernel
void test_matmul() {
    int N = 4; // Size of the NxN matrices

    // Allocate and initialize host matrices
    float h_A[N * N] = {1, 2, 3, 4,
                        5, 6, 7, 8,
                        9, 10, 11, 12,
                        13, 14, 15, 16};

    float h_B[N * N] = {1, 0, 0, 0,
                        0, 1, 0, 0,
                        0, 0, 1, 0,
                        0, 0, 0, 1};

    float h_C[N * N] = {0};

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_B, N * N * sizeof(float));
    hipMalloc(&d_C, N * N * sizeof(float));

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Define block size and grid size
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch the matrix multiplication kernel
    matmul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    std::cout << "Result matrix C:" << std::endl;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            std::cout << h_C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    test_matmul();
    return 0;
}
