#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "tensor.h"

template <typename Op>
struct IsMulOperation {
    static const bool value = false;
};

template <>
struct IsMulOperation<std::multiplies<float>> {
    static const bool value = true;
};

template <typename Op>
struct IsAddOrSubOperation {
    static const bool value = false;
};

template <>
struct IsAddOrSubOperation<std::plus<float>> {
    static const bool value = true;
};

template <>
struct IsAddOrSubOperation<std::minus<float>> {
    static const bool value = true;
};

__device__ void atomicmul(float* address, float val) {
    float old = *address;
    float assumed;
    do {
        assumed = old;
        old = atomicCAS((int*)address, __float_as_int(assumed), __float_as_int(assumed * val));
    } while (__float_as_int(*address) != __float_as_int(assumed));
}

__global__ void atomicMulKernel(float* data, float* values, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        atomicmul(&data[idx], values[idx]);
    }
}



void atomicMulTensor(Tensor<FLOAT32>& tensor, const Tensor<FLOAT32>& values) {
    if (tensor.shape != values.shape) {
        throw std::runtime_error("Shapes of the tensors do not match!");
    }

    int numElements = std::accumulate(tensor.shape.begin(), tensor.shape.end(), 1, std::multiplies<int>());

    float* d_tensorData;
    float* d_valuesData;
    hipMalloc(&d_tensorData, numElements * sizeof(float));
    hipMalloc(&d_valuesData, numElements * sizeof(float));

    hipMemcpy(d_tensorData, tensor.data(), numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_valuesData, values.data(), numElements * sizeof(float), hipMemcpyHostToDevice);
    
    int blockSize = 256;
    int numBlocks = (numElements + blockSize - 1) / blockSize;
    atomicMulKernel<<<numBlocks, blockSize>>>(d_tensorData, d_valuesData, numElements);
    hipMemcpy(tensor.data(), d_tensorData, numElements * sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(d_tensorData);
    hipFree(d_valuesData);

    hipDeviceSynchronize();
}

template <typename T>
__device__ void AtomicMul(T* address, T val) {
    if constexpr (std::is_same_v<T, float>) {
        atomicmul(address, val);
    } else {
        atomicmul(reinterpret_cast<float*>(address), static_cast<float>(val));
    }
}

template<typename T, typename Op>
__global__ void tensorOperationKernel(const T* a, const T* b, T* res, int num_elems, Op op) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elems) {
        if (IsMulOperation<Op>::value) {
            AtomicMul(&res[idx], op(a[idx], b[idx]));
        } else if (IsAddOrSubOperation<Op>::value) {
            atomicAdd(reinterpret_cast<int32_t*>(&res[idx]), static_cast<int32_t>(op(a[idx], b[idx])));
        } else {
            res[idx] = op(a[idx], b[idx]);
        }
    }
}


template <typename T, typename Op>
void tensorOperationCuda(const T* h_a, const T* h_b, T* h_result, int num_elems, Op op, int block_size) {
    int grid_size = (num_elems + block_size - 1) / block_size;

    T* d_a;
    T* d_b;
    T* d_result;

    hipMalloc(&d_a, num_elems * sizeof(T));
    hipMalloc(&d_b, num_elems * sizeof(T));
    hipMalloc(&d_result, num_elems * sizeof(T));

    hipMemcpy(d_a, h_a, num_elems * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, num_elems * sizeof(T), hipMemcpyHostToDevice);

    tensorOperationKernel<<<grid_size, block_size>>>(d_a, d_b, d_result, num_elems, op);
    hipDeviceSynchronize(); 

    hipMemcpy(h_result, d_result, num_elems * sizeof(T), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
}

__global__ void matmul_kernel(const float* A, const float* B, float* C, int m, int n, int p) {
    extern __shared__ float sharedMem[];

    float* Asub = sharedMem;
    float* Bsub = sharedMem + blockDim.y * blockDim.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0;

    for (int tile = 0; tile < (n + blockDim.x - 1) / blockDim.x; ++tile) {
        if (row < m && tile * blockDim.x + threadIdx.x < n) {
            Asub[threadIdx.y * blockDim.x + threadIdx.x] = A[row * n + tile * blockDim.x + threadIdx.x];
        } else {
            Asub[threadIdx.y * blockDim.x + threadIdx.x] = 0;
        }

        if (col < p && tile * blockDim.y + threadIdx.y < n) {
            Bsub[threadIdx.y * blockDim.x + threadIdx.x] = B[(tile * blockDim.y + threadIdx.y) * p + col];
        } else {
            Bsub[threadIdx.y * blockDim.x + threadIdx.x] = 0;
        }

        __syncthreads();

        for (int k = 0; k < blockDim.x; ++k) {
            sum += Asub[threadIdx.y * blockDim.x + k] * Bsub[k * blockDim.x + threadIdx.x]; 
        }

        __syncthreads();
    }

    if (row < m && col < p) {
        C[row * p + col] = sum;
    }
}


template <typename T>
void matmul_cuda(const T* h_A, const T* h_B, T* h_C, int m, int n, int p) {
    T* d_A;
    T* d_B;
    T* d_C;

    hipMalloc(&d_A, m * n * sizeof(T));
    hipMalloc(&d_B, n * p * sizeof(T));
    hipMalloc(&d_C, m * p * sizeof(T));

    hipMemcpy(d_A, h_A, m * n * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, n * p * sizeof(T), hipMemcpyHostToDevice);

    dim3 blockDim(16, 16); 
    dim3 gridDim((p + blockDim.x - 1) / blockDim.x, (m + blockDim.y - 1) / blockDim.y); 
    size_t sharedMemSize = 2 * blockDim.x * blockDim.y * sizeof(T);

    matmul_kernel<<<gridDim, blockDim, sharedMemSize>>>(reinterpret_cast<const float*>(d_A), 
                                                       reinterpret_cast<const float*>(d_B),
                                                       reinterpret_cast<float*>(d_C), m, n, p);

    hipDeviceSynchronize();
    hipMemcpy(h_C, d_C, m * p * sizeof(T), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

template void matmul_cuda<float>(const float*, const float*, float*, int, int, int);
template void matmul_cuda<int32_t>(const int32_t*, const int32_t*, int32_t*, int, int, int);
template void matmul_cuda<uint16_t>(const uint16_t*, const uint16_t*, uint16_t*, int, int, int);
template void matmul_cuda<uint8_t>(const uint8_t*, const uint8_t*, uint8_t*, int, int, int);
template void matmul_cuda<uint32_t>(const uint32_t*, const uint32_t*, uint32_t*, int, int, int);
template void matmul_cuda<int8_t>(const int8_t*, const int8_t*, int8_t*, int, int, int);


template void tensorOperationCuda<float, std::plus<float>>(const float*, const float*, float*, int, std::plus<float>, int);
template void tensorOperationCuda<float, std::minus<float>>(const float*, const float*, float*, int, std::minus<float>, int);
template void tensorOperationCuda<float, std::multiplies<float>>(const float*, const float*, float*, int, std::multiplies<float>, int);

template void tensorOperationCuda<int32_t, std::plus<int32_t>>(const int32_t*, const int32_t*, int32_t*, int, std::plus<int32_t>, int);
template void tensorOperationCuda<int32_t, std::minus<int32_t>>(const int32_t*, const int32_t*, int32_t*, int, std::minus<int32_t>, int);
template void tensorOperationCuda<int32_t, std::multiplies<int32_t>>(const int32_t*, const int32_t*, int32_t*, int, std::multiplies<int32_t>, int);

template void tensorOperationCuda<uint16_t, std::plus<uint16_t>>(const uint16_t*, const uint16_t*, uint16_t*, int, std::plus<uint16_t>, int);
template void tensorOperationCuda<uint16_t, std::minus<uint16_t>>(const uint16_t*, const uint16_t*, uint16_t*, int, std::minus<uint16_t>, int);
template void tensorOperationCuda<uint16_t, std::multiplies<uint16_t>>(const uint16_t*, const uint16_t*, uint16_t*, int, std::multiplies<uint16_t>, int);

template void tensorOperationCuda<uint8_t, std::plus<uint8_t>>(const uint8_t*, const uint8_t*, uint8_t*, int, std::plus<uint8_t>, int);
template void tensorOperationCuda<uint8_t, std::minus<uint8_t>>(const uint8_t*, const uint8_t*, uint8_t*, int, std::minus<uint8_t>, int);
template void tensorOperationCuda<uint8_t, std::multiplies<uint8_t>>(const uint8_t*, const uint8_t*, uint8_t*, int, std::multiplies<uint8_t>, int);

template void tensorOperationCuda<uint32_t, std::plus<uint32_t>>(const uint32_t*, const uint32_t*, uint32_t*, int, std::plus<uint32_t>, int);
template void tensorOperationCuda<uint32_t, std::minus<uint32_t>>(const uint32_t*, const uint32_t*, uint32_t*, int, std::minus<uint32_t>, int);
template void tensorOperationCuda<uint32_t, std::multiplies<uint32_t>>(const uint32_t*, const uint32_t*, uint32_t*, int, std::multiplies<uint32_t>, int);

template void tensorOperationCuda<int8_t, std::plus<int8_t>>(const int8_t*, const int8_t*, int8_t*, int, std::plus<int8_t>, int);
template void tensorOperationCuda<int8_t, std::minus<int8_t>>(const int8_t*, const int8_t*, int8_t*, int, std::minus<int8_t>, int);
template void tensorOperationCuda<int8_t, std::multiplies<int8_t>>(const int8_t*, const int8_t*, int8_t*, int, std::multiplies<int8_t>, int);
