#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "tensor.h"

template <typename Op>
struct IsMulOperation {
    static const bool value = false;
};

template <>
struct IsMulOperation<std::multiplies<float>> {
    static const bool value = true;
};

template <typename Op>
struct IsAddOrSubOperation {
    static const bool value = false;
};

template <>
struct IsAddOrSubOperation<std::plus<float>> {
    static const bool value = true;
};

template <>
struct IsAddOrSubOperation<std::minus<float>> {
    static const bool value = true;
};

__device__ void atomicmul(float* address, float val) {
    float old = *address;
    float assumed;
    do {
        assumed = old;
        old = atomicCAS((int*)address, __float_as_int(assumed), __float_as_int(assumed * val));
    } while (__float_as_int(*address) != __float_as_int(assumed));
}

__global__ void atomicMulKernel(float* data, float* values, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        atomicmul(&data[idx], values[idx]);
    }
}

__host__ void atomicMulTensor(Tensor<FLOAT32>& tensor, const Tensor<FLOAT32>& values) {
    if (tensor.shape != values.shape) {
        throw std::runtime_error("Shapes of the tensors do not match!");
    }

    int numElements = std::accumulate(tensor.shape.begin(), tensor.shape.end(), 1, std::multiplies<int>());

    float* tensorData = tensor.data();
    float* valuesData = values.data();
    
    std::cout<<"Within the kernel: "<<tensor<<std::endl;
    std::cout<<"Within the kernel, value: "<<values<<std::endl;
    int blockSize = 64;
    int numBlocks = (numElements + blockSize - 1) / blockSize;

    atomicMulKernel<<<numBlocks, blockSize>>>(tensorData, valuesData, numElements);
    
    std::cout<<"result: "<<std::endl;
    for(int i = 0; i<numElements; i++){
      std::cout<<tensorData[i]<<std::endl;
    }
    hipDeviceSynchronize();
}



template <typename T>
__device__ void AtomicMul(T* address, T val) {
    if constexpr (std::is_same_v<T, float>) {
        atomicmul(address, val);
    } else {
        atomicmul(reinterpret_cast<float*>(address), static_cast<float>(val));
    }
}

template<typename T, typename Op>
__global__ void tensorOperationKernel(const T* a, const T* b, T* res, int num_elems, Op op) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elems) {
        if (IsMulOperation<Op>::value) {
            AtomicMul(&res[idx], op(a[idx], b[idx]));
        } else if (IsAddOrSubOperation<Op>::value) {
            atomicAdd(reinterpret_cast<int32_t*>(&res[idx]), static_cast<int32_t>(op(a[idx], b[idx])));
        } else {
            res[idx] = op(a[idx], b[idx]);
        }
    }
}

template <typename T, typename Op>
void tensorOperationCuda(const T* a, const T* b, T* result, int num_elems, Op op, int block_size) {
    int grid_size = (num_elems + block_size - 1) / block_size;
    tensorOperationKernel<<<grid_size, block_size>>>(a, b, result, num_elems, op);
    hipDeviceSynchronize(); 

    // Print results
    T* host_result = new T[num_elems];
    hipMemcpy(host_result, result, num_elems * sizeof(T), hipMemcpyDeviceToHost);

    std::cout << "Result: ";
    for (int i = 0; i < num_elems; ++i) {
        std::cout << host_result[i] << " ";
    }
    std::cout << std::endl;

    delete[] host_result;
}

__global__ void matmul_kernel(const float* A, const float* B, float* C, int m, int n, int p) {
    extern __shared__ float sharedMem[];

    float* Asub = sharedMem;
    float* Bsub = sharedMem + blockDim.y * blockDim.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0;

    printf("Thread [%d, %d]: Initial sum = %f\n", row, col, sum);

    for (int tile = 0; tile < (n + blockDim.x - 1) / blockDim.x; ++tile) {
        if (row < m && tile * blockDim.x + threadIdx.x < n) {
            Asub[threadIdx.y * blockDim.x + threadIdx.x] = A[row * n + tile * blockDim.x + threadIdx.x];
        } else {
            Asub[threadIdx.y * blockDim.x + threadIdx.x] = 0;
        }

        if (col < p && tile * blockDim.y + threadIdx.y < n) {
            Bsub[threadIdx.y * blockDim.x + threadIdx.x] = B[(tile * blockDim.y + threadIdx.y) * p + col];
        } else {
            Bsub[threadIdx.y * blockDim.x + threadIdx.x] = 0;
        }

        __syncthreads();

        for (int k = 0; k < blockDim.x; ++k) {
            sum += Asub[threadIdx.y * blockDim.x + k] * Bsub[k * blockDim.x + threadIdx.x]; 
        }

        __syncthreads();
    }

    if (row < m && col < p) {
        C[row * p + col] = sum;
    }
}

template <typename T>
void matmul_cuda(const T* A, const T* B, T* C, int m, int n, int p) {
    // Print inputs
    std::cout << "Input A (Matrix A):" << std::endl;
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            std::cout << A[i * n + j] << " ";
        }
        std::cout << std::endl;
    }

    std::cout << "Input B (Matrix B):" << std::endl;
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < p; ++j) {
            std::cout << B[i * p + j] << " ";
        }
        std::cout << std::endl;
    }

    std::cout << "Matrix Dimensions:" << std::endl;
    std::cout << "m: " << m << ", n: " << n << ", p: " << p << std::endl;

    dim3 blockDim(16, 16); 
    dim3 gridDim((p + blockDim.x - 1) / blockDim.x, (m + blockDim.y - 1) / blockDim.y); 
    size_t sharedMemSize = 2 * blockDim.x * blockDim.y * sizeof(T);

    matmul_kernel<<<gridDim, blockDim, sharedMemSize>>>(reinterpret_cast<const float*>(A), 
                                                       reinterpret_cast<const float*>(B),
                                                       reinterpret_cast<float*>(C), m, n, p);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    }
    hipDeviceSynchronize();

    // Print output
    std::cout << "Output C (Resulting Matrix):" << std::endl;
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < p; ++j) {
            std::cout << C[i * p + j] << " ";
        }
        std::cout << std::endl;
    }
}


template void matmul_cuda<float>(const float*, const float*, float*, int, int, int);
template void matmul_cuda<int32_t>(const int32_t*, const int32_t*, int32_t*, int, int, int);
template void matmul_cuda<uint16_t>(const uint16_t*, const uint16_t*, uint16_t*, int, int, int);
template void matmul_cuda<uint8_t>(const uint8_t*, const uint8_t*, uint8_t*, int, int, int);
template void matmul_cuda<uint32_t>(const uint32_t*, const uint32_t*, uint32_t*, int, int, int);
template void matmul_cuda<int8_t>(const int8_t*, const int8_t*, int8_t*, int, int, int);


template void tensorOperationCuda<float, std::plus<float>>(const float*, const float*, float*, int, std::plus<float>, int);
template void tensorOperationCuda<float, std::minus<float>>(const float*, const float*, float*, int, std::minus<float>, int);
template void tensorOperationCuda<float, std::multiplies<float>>(const float*, const float*, float*, int, std::multiplies<float>, int);

template void tensorOperationCuda<int32_t, std::plus<int32_t>>(const int32_t*, const int32_t*, int32_t*, int, std::plus<int32_t>, int);
template void tensorOperationCuda<int32_t, std::minus<int32_t>>(const int32_t*, const int32_t*, int32_t*, int, std::minus<int32_t>, int);
template void tensorOperationCuda<int32_t, std::multiplies<int32_t>>(const int32_t*, const int32_t*, int32_t*, int, std::multiplies<int32_t>, int);

template void tensorOperationCuda<uint16_t, std::plus<uint16_t>>(const uint16_t*, const uint16_t*, uint16_t*, int, std::plus<uint16_t>, int);
template void tensorOperationCuda<uint16_t, std::minus<uint16_t>>(const uint16_t*, const uint16_t*, uint16_t*, int, std::minus<uint16_t>, int);
template void tensorOperationCuda<uint16_t, std::multiplies<uint16_t>>(const uint16_t*, const uint16_t*, uint16_t*, int, std::multiplies<uint16_t>, int);

template void tensorOperationCuda<uint8_t, std::plus<uint8_t>>(const uint8_t*, const uint8_t*, uint8_t*, int, std::plus<uint8_t>, int);
template void tensorOperationCuda<uint8_t, std::minus<uint8_t>>(const uint8_t*, const uint8_t*, uint8_t*, int, std::minus<uint8_t>, int);
template void tensorOperationCuda<uint8_t, std::multiplies<uint8_t>>(const uint8_t*, const uint8_t*, uint8_t*, int, std::multiplies<uint8_t>, int);

template void tensorOperationCuda<uint32_t, std::plus<uint32_t>>(const uint32_t*, const uint32_t*, uint32_t*, int, std::plus<uint32_t>, int);
template void tensorOperationCuda<uint32_t, std::minus<uint32_t>>(const uint32_t*, const uint32_t*, uint32_t*, int, std::minus<uint32_t>, int);
template void tensorOperationCuda<uint32_t, std::multiplies<uint32_t>>(const uint32_t*, const uint32_t*, uint32_t*, int, std::multiplies<uint32_t>, int);

template void tensorOperationCuda<int8_t, std::plus<int8_t>>(const int8_t*, const int8_t*, int8_t*, int, std::plus<int8_t>, int);
template void tensorOperationCuda<int8_t, std::minus<int8_t>>(const int8_t*, const int8_t*, int8_t*, int, std::minus<int8_t>, int);
template void tensorOperationCuda<int8_t, std::multiplies<int8_t>>(const int8_t*, const int8_t*, int8_t*, int, std::multiplies<int8_t>, int);
