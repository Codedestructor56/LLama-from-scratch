#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "tensor.h"

template <typename Op>
struct IsMulOperation {
    static const bool value = false;
};

template <>
struct IsMulOperation<std::multiplies<float>> {
    static const bool value = true;
};

template <typename Op>
struct IsAddOrSubOperation {
    static const bool value = false;
};

template <>
struct IsAddOrSubOperation<std::plus<float>> {
    static const bool value = true;
};

template <>
struct IsAddOrSubOperation<std::minus<float>> {
    static const bool value = true;
};

__device__ void atomicmul(float* address, float val) {
    float old = *address;
    float assumed;
    do {
        assumed = old;
        old = atomicCAS((int*)address, __float_as_int(assumed), __float_as_int(assumed * val));
    } while (__float_as_int(*address) != __float_as_int(assumed));
}

template <typename T>
__device__ void AtomicMul(T* address, T val) {
    if constexpr (std::is_same_v<T, float>) {
        atomicmul(address, val);
    } else {
        atomicmul(reinterpret_cast<float*>(address), static_cast<float>(val));
    }
}

template<typename T, typename Op>
__global__ void tensorOperationKernel(const T* a, const T* b, T* res, int num_elems, Op op) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elems) {
        if (IsMulOperation<Op>::value) {
            AtomicMul(&res[idx], op(a[idx], b[idx]));
        } else if (IsAddOrSubOperation<Op>::value) {
            atomicAdd(reinterpret_cast<int32_t*>(&res[idx]), static_cast<int32_t>(op(a[idx], b[idx])));
        } else {
            res[idx] = op(a[idx], b[idx]);
        }
    }
}

template <typename T, typename Op>
void tensorOperationCuda(const T* a, const T* b, T* result, int num_elems, Op op, int block_size) {
    int grid_size = (num_elems + block_size - 1) / block_size;
    tensorOperationKernel<<<grid_size, block_size>>>(a, b, result, num_elems, op);
    hipDeviceSynchronize(); 
}

__global__ void matmul_kernel(const float* A, const float* B, float* C, int m, int n, int p) {
    extern __shared__ float sharedMem[];
 
    float* Asub = sharedMem;
    float* Bsub = sharedMem + blockDim.y * blockDim.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
 
    float sum = 0;

    for (int tile = 0; tile < (n + blockDim.x - 1) / blockDim.x; ++tile) {
        if (row < m && tile * blockDim.x + threadIdx.x < n) {
            Asub[threadIdx.y * blockDim.x + threadIdx.x] = A[row * n + tile * blockDim.x + threadIdx.x];
        } else {
            Asub[threadIdx.y * blockDim.x + threadIdx.x] = 0;
        }

        if (col < p && tile * blockDim.y + threadIdx.y < n) {
            Bsub[threadIdx.y * blockDim.x + threadIdx.x] = B[(tile * blockDim.y + threadIdx.y) * p + col];
        } else {
            Bsub[threadIdx.y * blockDim.x + threadIdx.x] = 0;
        }

        __syncthreads(); 
        for (int k = 0; k < blockDim.x; ++k) {
            sum += Asub[threadIdx.y * blockDim.x + k] * Bsub[k * blockDim.x + threadIdx.x];
        }

        __syncthreads(); 
    }

    if (row < m && col < p) {
        C[row * p + col] = sum;
    }
}

template <typename T>
void matmul_cuda(const T* A, const T* B, T* C, int m, int n, int p) {
    dim3 blockDim(16, 16); 
    dim3 gridDim((p + blockDim.x - 1) / blockDim.x, (m + blockDim.y - 1) / blockDim.y); 
    size_t sharedMemSize = 2 * blockDim.x * blockDim.y * sizeof(T);

    matmul_kernel<<<gridDim, blockDim, sharedMemSize>>>(reinterpret_cast<const float*>(A), reinterpret_cast<const float*>(B),
        reinterpret_cast<float*>(C), m, n, p);
    
    C = reinterpret_cast<T*>(C);
    hipDeviceSynchronize();
}

template void matmul_cuda<float>(const float*, const float*, float*, int, int, int);
template void matmul_cuda<int32_t>(const int32_t*, const int32_t*, int32_t*, int, int, int);
template void matmul_cuda<uint16_t>(const uint16_t*, const uint16_t*, uint16_t*, int, int, int);
template void matmul_cuda<uint8_t>(const uint8_t*, const uint8_t*, uint8_t*, int, int, int);
template void matmul_cuda<uint32_t>(const uint32_t*, const uint32_t*, uint32_t*, int, int, int);
template void matmul_cuda<int8_t>(const int8_t*, const int8_t*, int8_t*, int, int, int);


template void tensorOperationCuda<float, std::plus<float>>(const float*, const float*, float*, int, std::plus<float>, int);
template void tensorOperationCuda<float, std::minus<float>>(const float*, const float*, float*, int, std::minus<float>, int);
template void tensorOperationCuda<float, std::multiplies<float>>(const float*, const float*, float*, int, std::multiplies<float>, int);

template void tensorOperationCuda<int32_t, std::plus<int32_t>>(const int32_t*, const int32_t*, int32_t*, int, std::plus<int32_t>, int);
template void tensorOperationCuda<int32_t, std::minus<int32_t>>(const int32_t*, const int32_t*, int32_t*, int, std::minus<int32_t>, int);
template void tensorOperationCuda<int32_t, std::multiplies<int32_t>>(const int32_t*, const int32_t*, int32_t*, int, std::multiplies<int32_t>, int);

template void tensorOperationCuda<uint16_t, std::plus<uint16_t>>(const uint16_t*, const uint16_t*, uint16_t*, int, std::plus<uint16_t>, int);
template void tensorOperationCuda<uint16_t, std::minus<uint16_t>>(const uint16_t*, const uint16_t*, uint16_t*, int, std::minus<uint16_t>, int);
template void tensorOperationCuda<uint16_t, std::multiplies<uint16_t>>(const uint16_t*, const uint16_t*, uint16_t*, int, std::multiplies<uint16_t>, int);

template void tensorOperationCuda<uint8_t, std::plus<uint8_t>>(const uint8_t*, const uint8_t*, uint8_t*, int, std::plus<uint8_t>, int);
template void tensorOperationCuda<uint8_t, std::minus<uint8_t>>(const uint8_t*, const uint8_t*, uint8_t*, int, std::minus<uint8_t>, int);
template void tensorOperationCuda<uint8_t, std::multiplies<uint8_t>>(const uint8_t*, const uint8_t*, uint8_t*, int, std::multiplies<uint8_t>, int);

template void tensorOperationCuda<uint32_t, std::plus<uint32_t>>(const uint32_t*, const uint32_t*, uint32_t*, int, std::plus<uint32_t>, int);
template void tensorOperationCuda<uint32_t, std::minus<uint32_t>>(const uint32_t*, const uint32_t*, uint32_t*, int, std::minus<uint32_t>, int);
template void tensorOperationCuda<uint32_t, std::multiplies<uint32_t>>(const uint32_t*, const uint32_t*, uint32_t*, int, std::multiplies<uint32_t>, int);

template void tensorOperationCuda<int8_t, std::plus<int8_t>>(const int8_t*, const int8_t*, int8_t*, int, std::plus<int8_t>, int);
template void tensorOperationCuda<int8_t, std::minus<int8_t>>(const int8_t*, const int8_t*, int8_t*, int, std::minus<int8_t>, int);
template void tensorOperationCuda<int8_t, std::multiplies<int8_t>>(const int8_t*, const int8_t*, int8_t*, int, std::multiplies<int8_t>, int);
