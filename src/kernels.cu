#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "tensor.h"

template <typename Op>
struct IsMulOperation {
    static const bool value = false;
};

template <>
struct IsMulOperation<std::multiplies<float>> {
    static const bool value = true;
};

template <typename Op>
struct IsAddOrSubOperation {
    static const bool value = false;
};

template <>
struct IsAddOrSubOperation<std::plus<float>> {
    static const bool value = true;
};

template <>
struct IsAddOrSubOperation<std::minus<float>> {
    static const bool value = true;
};

template <typename T>
__device__ void atomicMul(T* address, T val) {
    if constexpr (std::is_same_v<T, float>) {
        atomicMul(address, val);
    } else {
        *address *= val;
    }
}

template<typename T, typename Op>
__global__ void tensorOperationKernel(const T* a, const T* b, T* res, int num_elems, Op op) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elems) {
        if (IsMulOperation<Op>::value) {
            atomicMul(&res[idx], op(a[idx], b[idx]));
        } else if (IsAddOrSubOperation<Op>::value) {
            atomicAdd(reinterpret_cast<int32_t*>(&res[idx]), static_cast<int32_t>(op(a[idx], b[idx])));
        } else {
            res[idx] = op(a[idx], b[idx]);
        }
    }
}

template <typename T, typename Op>
void tensorOperationCuda(const T* a, const T* b, T* result, int num_elems, Op op, int block_size) {
    int grid_size = (num_elems + block_size - 1) / block_size;
    tensorOperationKernel<<<grid_size, block_size>>>(a, b, result, num_elems, op);
    hipDeviceSynchronize(); 
}

template void tensorOperationCuda<float, std::plus<float>>(const float*, const float*, float*, int, std::plus<float>, int);
template void tensorOperationCuda<float, std::minus<float>>(const float*, const float*, float*, int, std::minus<float>, int);
template void tensorOperationCuda<float, std::multiplies<float>>(const float*, const float*, float*, int, std::multiplies<float>, int);

template void tensorOperationCuda<int32_t, std::plus<int32_t>>(const int32_t*, const int32_t*, int32_t*, int, std::plus<int32_t>, int);
template void tensorOperationCuda<int32_t, std::minus<int32_t>>(const int32_t*, const int32_t*, int32_t*, int, std::minus<int32_t>, int);
template void tensorOperationCuda<int32_t, std::multiplies<int32_t>>(const int32_t*, const int32_t*, int32_t*, int, std::multiplies<int32_t>, int);

template void tensorOperationCuda<uint16_t, std::plus<uint16_t>>(const uint16_t*, const uint16_t*, uint16_t*, int, std::plus<uint16_t>, int);
template void tensorOperationCuda<uint16_t, std::minus<uint16_t>>(const uint16_t*, const uint16_t*, uint16_t*, int, std::minus<uint16_t>, int);
template void tensorOperationCuda<uint16_t, std::multiplies<uint16_t>>(const uint16_t*, const uint16_t*, uint16_t*, int, std::multiplies<uint16_t>, int);

template void tensorOperationCuda<uint8_t, std::plus<uint8_t>>(const uint8_t*, const uint8_t*, uint8_t*, int, std::plus<uint8_t>, int);
template void tensorOperationCuda<uint8_t, std::minus<uint8_t>>(const uint8_t*, const uint8_t*, uint8_t*, int, std::minus<uint8_t>, int);
template void tensorOperationCuda<uint8_t, std::multiplies<uint8_t>>(const uint8_t*, const uint8_t*, uint8_t*, int, std::multiplies<uint8_t>, int);

template void tensorOperationCuda<uint32_t, std::plus<uint32_t>>(const uint32_t*, const uint32_t*, uint32_t*, int, std::plus<uint32_t>, int);
template void tensorOperationCuda<uint32_t, std::minus<uint32_t>>(const uint32_t*, const uint32_t*, uint32_t*, int, std::minus<uint32_t>, int);
template void tensorOperationCuda<uint32_t, std::multiplies<uint32_t>>(const uint32_t*, const uint32_t*, uint32_t*, int, std::multiplies<uint32_t>, int);

template void tensorOperationCuda<int8_t, std::plus<int8_t>>(const int8_t*, const int8_t*, int8_t*, int, std::plus<int8_t>, int);
template void tensorOperationCuda<int8_t, std::minus<int8_t>>(const int8_t*, const int8_t*, int8_t*, int, std::minus<int8_t>, int);
template void tensorOperationCuda<int8_t, std::multiplies<int8_t>>(const int8_t*, const int8_t*, int8_t*, int, std::multiplies<int8_t>, int);
