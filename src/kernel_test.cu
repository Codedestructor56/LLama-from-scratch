#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "tensor.h"

__global__ void atomicMulKernel(float* data, float* values, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        //atomicmul(&data[idx], values[idx]);
        data[idx] *= values[idx];
    }
}

void atomicMulTensor(Tensor<FLOAT32>& tensor, const Tensor<FLOAT32>& values) {
    if (tensor.shape != values.shape) {
        throw std::runtime_error("Shapes of the tensors do not match!");
    }

    int numElements = std::accumulate(tensor.shape.begin(), tensor.shape.end(), 1, std::multiplies<int>());

    float* tensorData = tensor.data();
    float* valuesData = values.data();
    
    std::cout<<"Within the kernel: "<<tensor<<std::endl;
    std::cout<<"Within the kernel, value: "<<values<<std::endl;
    int blockSize = 256;
    int numBlocks = (numElements + blockSize - 1) / blockSize;

    atomicMulKernel<<<numBlocks, blockSize>>>(tensorData, valuesData, numElements);
hipError_t err = hipGetLastError(); 
    if (err != hipSuccess) {
    std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
} 
  hipDeviceSynchronize();
  std::cout<<"result: "<<std::endl;
    for(int i = 0; i<numElements; i++){
      std::cout<<tensorData[i]<<std::endl;
    }

}


